#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>

#define BLOCK_SIZE 256
#define BLOCK_COUNT 216
 
#define CHAR_START 33
#define CHAR_END 123
#define CHAR2_START 48
#define CHAR2_END 112

typedef unsigned long long  crc_t;

// массив для хранения коэффициентов полинома
__constant__ crc_t crc_table[256];
static crc_t crc_table_cpu[256];

__constant__ char hex_digit[17] = "0123456789abcdef";
char hex_digit_cpu[17] = "0123456789abcdef";

// заполнение таблицы коэффициентов полинома
void gen_table(crc_t* tbl) {
	crc_t crc64Poly = 0xC96C5795D7870F42;
	for (unsigned i = 0; i < 256; ++i) {
		crc_t _crc = i;
		for (unsigned j = 0; j < 8; ++j) {
			_crc = _crc & 1 ? (_crc >> 1) ^ crc64Poly : _crc >> 1;
		}
		tbl[i] = _crc;
	}
}

// расчет crc64 на CPU
// параметры: crc - начальное значение, 
// data - строка, для которой выполняется расчет, 
// data_len - длина строки
crc_t crc_update_cpu(crc_t crc, const void* data, size_t data_len)
{
	const unsigned char* d = (const unsigned char*)data;
	unsigned int tbl_idx;

	while (data_len--) {
		tbl_idx = (crc ^ *d) & 0xff;
		crc = (crc_table_cpu[tbl_idx] ^ (crc >> 8));
		d++;
	}
	return crc;
}

// расчет crc64 на GPU
// параметры: crc - начальное значение, 
// data - строка, для которой выполняется расчет, 
// data_len - длина строки
__device__ inline crc_t crc_update_gpu(crc_t crc, const void* data, size_t data_len) {
	const unsigned char* d = (const unsigned char*)data;
	unsigned int tbl_idx;

	while (data_len--) {
		tbl_idx = (crc ^ *d) & 0xff;
		crc = (crc_table[tbl_idx] ^ (crc >> 8));
		d++;
	}
	return crc;
}


// поиск минимального crc64
// к строке добавляется три символа, сформированных из номера блока и номера нити
// и четыре символа, сформированных перебором в цикле
__global__ void kernel_CRC64_min(crc_t crc_pref, crc_t* seq, crc_t* res, crc_t* val) {
	unsigned tx = threadIdx.x;
	unsigned bx = blockIdx.x;
	crc_t tnum = blockIdx.x * BLOCK_SIZE + tx;

	__shared__ crc_t crc_table_k[256];
	__shared__ crc_t th_min[BLOCK_SIZE];
	__shared__ crc_t th_str[BLOCK_SIZE];

	// копирование таблицы коэффициентов полинома в разделяемую память
	if (tx == 0) {
		for (unsigned i = 0; i < 256; ++i) {
			crc_table_k[i] = crc_table[i];
		}
	}
	__syncthreads();

	// добавление трех символов, получаемых из номера блока и номера нити
	crc_t crc0 = crc_pref;
	crc_t ch3 = ((tnum >> 12) & 0x3f) + CHAR2_START;
	crc_t ch2 = ((tnum >> 6) & 0x3f) + CHAR2_START;
	crc_t ch1 = ((tnum) & 0x3f) + CHAR2_START;

	crc0 = crc_table_k[(crc0 ^ ch3) & 0xff] ^ (crc0 >> 8);
	crc0 = crc_table_k[(crc0 ^ ch2) & 0xff] ^ (crc0 >> 8);
	crc0 = crc_table_k[(crc0 ^ ch1) & 0xff] ^ (crc0 >> 8);

	crc_t crc_min = 0;
	crc_t str;
	th_min[tx] = 0;

	// добавление четырех символов из цикла перебора и поиск минимального значения
	for (crc_t i1 = CHAR_START; i1 < CHAR_END; ++i1) {
		crc_t cur_crc1 = (crc_table_k[(crc0 ^ i1) & 0xff] ^ (crc0 >> 8));
		for (crc_t i2 = CHAR_START; i2 < CHAR_END; ++i2) {
			crc_t cur_crc2 = (crc_table_k[(cur_crc1 ^ i2) & 0xff] ^ (cur_crc1 >> 8));
			for (crc_t i3 = CHAR_START; i3 < CHAR_END; ++i3) {
				crc_t cur_crc3 = (crc_table_k[(cur_crc2 ^ i3) & 0xff] ^ (cur_crc2 >> 8));
				for (crc_t i4 = CHAR_START; i4 < CHAR_END; ++i4) {
					crc_t cur_crc4 = (crc_table_k[(cur_crc3 ^ i4) & 0xff] ^ (cur_crc3 >> 8));
					if (cur_crc4 > th_min[tx]) {
						th_str[tx] = (ch3 << 48) | (ch2 << 40) | (ch1 << 32) | (i1 << 24) | (i2 << 16) | (i3 << 8) | (i4);
						th_min[tx] = cur_crc4;
					}
				}
			}
		}
	}

	__syncthreads();
	if (tx == 0) {
		crc_min = th_min[0];
		for (unsigned i = 0; i < BLOCK_SIZE; ++i) {
			if (th_min[i] >= crc_min) {
				crc_min = th_min[i];
				str = th_str[i];
			}
		}
		res[bx] = crc_min;
		seq[bx] = str;
	}

}

// функция для перевода в строку
void seq_to_str(crc_t seq, char* str) {
	for (unsigned i = 0; i < 7; ++i) {
		str[6 - i] = (char)((seq >> (i * 8)) & 0xff);
		str[7] = 0;
	}
}

// переписанная функция kernel_CRC64_min для CPU для сравнения времени работы 
void find_CRC64_min_CPU(crc_t crc_pref, crc_t* val, unsigned char* data) {
	data[0] = CHAR2_START;
	data[1] = CHAR2_START;
	data[7] = 0;
	crc_pref = (crc_table_cpu[(crc_pref ^ CHAR2_START) & 0xff] ^ (crc_pref >> 8));
	crc_pref = (crc_table_cpu[(crc_pref ^ CHAR2_START) & 0xff] ^ (crc_pref >> 8));
	crc_t crc_min = 0LL;
	for (crc_t i1 = CHAR2_START; i1 < CHAR2_END; ++i1) {
		crc_t cur_crc1 = (crc_table_cpu[(crc_pref ^ i1) & 0xff] ^ (crc_pref >> 8));
		for (crc_t i2 = CHAR_START; i2 < CHAR_END; ++i2) {
			crc_t cur_crc2 = (crc_table_cpu[(cur_crc1 ^ i2) & 0xff] ^ (cur_crc1 >> 8));
			for (crc_t i3 = CHAR_START; i3 < CHAR_END; ++i3) {
				crc_t cur_crc3 = (crc_table_cpu[(cur_crc2 ^ i3) & 0xff] ^ (cur_crc2 >> 8));
				for (crc_t i4 = CHAR_START; i4 < CHAR_END; ++i4) {
					crc_t cur_crc4 = (crc_table_cpu[(cur_crc3 ^ i4) & 0xff] ^ (cur_crc3 >> 8));
					for (crc_t i5 = CHAR_START; i5 < CHAR_END; ++i5) {
						crc_t cur_crc5 = (crc_table_cpu[(cur_crc4 ^ i5) & 0xff] ^ (cur_crc4 >> 8));
							if (cur_crc5 > crc_min) {
								crc_min = cur_crc5;
								data[2] = (uint8_t)i1;
								data[3] = (uint8_t)i2;
								data[4] = (uint8_t)i3;
								data[5] = (uint8_t)i4;
								data[6] = (uint8_t)i5;
								*val = ~cur_crc5;
							}
					}
				}
			}
		}
	}
}


// организация вычислений на GPU
void find_min_CRC64_GPU(crc_t crc_pref, crc_t* res, crc_t* seq) {
	const int size_d = 20;

	gen_table(crc_table_cpu);
	hipMemcpyToSymbol(HIP_SYMBOL(crc_table), &crc_table_cpu, 256 * sizeof(crc_t), 0, hipMemcpyHostToDevice);

	crc_t d[size_d];
	crc_t cpu_seq[BLOCK_COUNT];
	crc_t cpu_res[BLOCK_COUNT];

	crc_t* dev_d = 0;
	crc_t* dev_seq = 0;
	crc_t* dev_res = 0;

	hipMalloc((void**)&dev_d, size_d * sizeof(crc_t));
	hipMalloc((void**)&dev_seq, BLOCK_COUNT * sizeof(crc_t));
	hipMalloc((void**)&dev_res, BLOCK_COUNT * sizeof(crc_t));

	kernel_CRC64_min << < BLOCK_COUNT, BLOCK_SIZE >> > (crc_pref, dev_seq, dev_res, dev_d);

	hipMemcpy(d, dev_d, size_d * sizeof(crc_t), hipMemcpyDeviceToHost);
	hipMemcpy(cpu_seq, dev_seq, BLOCK_COUNT * sizeof(crc_t), hipMemcpyDeviceToHost);
	hipMemcpy(cpu_res, dev_res, BLOCK_COUNT * sizeof(crc_t), hipMemcpyDeviceToHost);

	crc_t res_min = cpu_res[0], seq_min = 0;
	for (unsigned i = 0; i < BLOCK_COUNT; ++i) {
		if (cpu_res[i] >= res_min) {
			res_min = cpu_res[i];
			seq_min = cpu_seq[i];
		}
	}

	*res = res_min;
	*seq = seq_min;

	hipFree(dev_d);
	hipFree(dev_seq);
	hipFree(dev_res);
}


int main(int argc, char** argv) {

	// обработка аргументов командной строки
	if (argc < 2) {
		printf("err %d\n", argc);
		return 1;
	}

	// при вызове с аргументом check
	// возвращает слово count, количество работающих GPU
	if (strcmp(argv[1], "check") == 0) { 
		int deviceCount = 0;
		hipGetDeviceCount(&deviceCount);
		printf("count %d", deviceCount);
	}
	// при вызове с аргументами calc и crc_pref,
	// где crc_pref - начальное значение crc64,
	// возвращает слово crc,
	// строку, которой соотвествует минимальное значение crc64,
	// минимальное значение crc64, 
	// суммарное количество итераций на всех нитях и блоках
	else if (strcmp(argv[1], "calc") == 0) {
		crc_t res, seq, count = 1;
		count = count * 90 * 90 * 90 * 90 * BLOCK_SIZE * BLOCK_COUNT;
		find_min_CRC64_GPU(~strtoull(argv[2], NULL, 10), &res, &seq);
		char str3[16];
		seq_to_str(seq, str3);
		printf("crc %s %016llx %lld", str3, ~res, count);
	}
	else {
		printf("err %d\n", argc);
	}

	return 0;
}